#include "hip/hip_runtime.h"
#include "common.h"

#include <algorithm>

#include <hip/hip_runtime.h>

#define A(r, c, h) A[(h)*(c) + (r)]
//#define A(r, c) A[n*(c) + (r)]

#define HANDLE_ERROR(err) HandleError(err, __FILE__, __LINE__)
void HandleError(hipError_t err, const char* file, int line)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(1);
	}
}

void syncDevice(float* hostP, float* deviceP, int m, int n)
{
	HANDLE_ERROR(hipMemcpy(deviceP, hostP, m*n*sizeof(float), hipMemcpyHostToDevice));
}

void syncHost(float* hostP, float* deviceP, int m, int n)
{
	HANDLE_ERROR(hipMemcpy(hostP, deviceP, m*n*sizeof(float), hipMemcpyDeviceToHost));
}

void printDeviceMatrix(int m, int n, float* A, float* Ad)
{
	syncHost(A, Ad, m, n);
	printMatrix(m, n, A);
}

__device__ void printSubMatrix(float* A, int m, int n, int h)
{
	int id = blockDim.x*threadIdx.y + threadIdx.x;
	
	if (id == 0)
	{	
		for (int j = 0; j < m; j++)
		{
			for (int i = 0; i < n; i++)
				printf("%7.2f", A[i*h + j]);
			printf("\n");
		}
		printf("\n");
	}
}

__device__ void copySubMatrix(float* src, float* dest, int m, int n, int srcH, int destH)
{
	if (threadIdx.x < m && threadIdx.y < n)
		dest[threadIdx.y*destH + threadIdx.x] = src[threadIdx.y*srcH + threadIdx.x];
}

__global__ void factorizeKernel(int m, int n, int h, float* A)
{
	extern __shared__ float C[];
	
	float* B = A + (blockIdx.x + 1)*n;
	int Cm = min(n, m - (blockIdx.x + 1)*n);
	m = n + Cm;
	
	copySubMatrix(A, C, n, n, h, m);
	copySubMatrix(B, C + n, Cm, n, h, m);
	
	__syncthreads();
	
	int id = blockDim.x*threadIdx.y + threadIdx.x;
	
	for (int i = 0; i < min(m - 1, n); ++i)
	{
    	float tmp = 1/C[i*m + i];
    	
    	int l = i + 1 + id;
    	if (l < m)
    		C[i*m + l] *= tmp;
    		
    	__syncthreads();
        
        /*#pragma omp for
		for (int l = i + 1; l < m; ++l)
			A(l, i, h) *= tmp;*/

		for (int k = i + 1 + threadIdx.y; k < n; k += blockDim.y)
			for (int l = i + 1 + threadIdx.x; l < m; l += blockDim.x)
				C[k*m + l] -= C[i*m + l]*C[k*m + i];

		__syncthreads();

		/*#pragma omp for
		for (int k = i + 1; k < n; ++k)
			for (int l = i + 1; l < m; ++l)
				A(l, k, h) -= A(l, i, h)*A(i, k, h);*/
	}
    
    //if (blockIdx.x == 0) printSubMatrix(C, m, n, m); __syncthreads();
    
	if (blockIdx.x == 0)
		copySubMatrix(C, A, n, n, m, h);
	copySubMatrix(C + n, B, Cm, n, m, h);
}

void factorize(int m, int n, int h, float* Ad)
{
	dim3 grid(max((m - 1)/n, 1));
	dim3 block(n, n);
	int size = 2*n*n*sizeof(float);
	
	factorizeKernel<<<grid, block, size>>>(m, n, h, Ad);
}

__global__ void updateRightKernel(int m, int n, int h, float* A)
{
	extern __shared__ float C[];
	
	float* B = A + (blockIdx.x + 1)*m*h;
	int Dn = min(m, n - (blockIdx.x + 1)*m);
	float* D = C + m*m;
	
	copySubMatrix(A, C, m, m, h, m);
	copySubMatrix(B, D, m, Dn, h, m);
	
	__syncthreads();
	
	if (threadIdx.y < Dn)
	{
		for (int k = 0; k < m - 1; ++k)
		{
			//if (blockIdx.x == 0) printSubMatrix(D, m, Dn, m); __syncthreads();
			
			int l = k + 1 + threadIdx.x;
			if (l < m)
				D[threadIdx.y*m + l] -= D[threadIdx.y*m + k]*C[k*m + l];
			
			__syncthreads();
		}
	}
	
	/*#pragma omp for
	for (int j = m; j < n; ++j)
	{
		for (int k = 0; k < m - 1; ++k)
			for (int l = k + 1; l < m; ++l)
				A(l, j, h) -= A(k, j, h)*A(l, k, h);
	}*/
    
    //if (blockIdx.x == 0) printSubMatrix(D, m, Dn, m); __syncthreads();
    
	copySubMatrix(D, B, m, Dn, m, h);
}

void updateRight(int m, int n, int h, float* Ad)
{
	dim3 grid((n - 1)/m);
	dim3 block(m, m);
	int size = 2*m*m*sizeof(float);
	
	updateRightKernel<<<grid, block, size>>>(m, n, h, Ad);
}

__global__ void updateDownKernel(int n, const int k, int h, float* A)
{
	extern __shared__ float B[];
	
	float* C = B + k*k;
	float* D = B + 2*k*k;
	int Cm = min(k, n - (blockIdx.x + 1)*k);
	int Dn = min(k, n - (blockIdx.y + 1)*k);
	
	float* Bglobal = A + (blockIdx.x + 1)*k + (blockIdx.y + 1)*k*h;
	float* Cglobal = A + (blockIdx.x + 1)*k;
	float* Dglobal = A + (blockIdx.y + 1)*k*h;
	
	copySubMatrix(Bglobal, B, Cm, Dn, h, k);
	copySubMatrix(Cglobal, C, Cm, k, h, k);
	copySubMatrix(Dglobal, D, k, Dn, h, k);
	
	__syncthreads();
	
	int x = threadIdx.x;
	int y = threadIdx.y;
	
	if (x < Cm && y < Dn)
	{
		D += y*k;
		C += x;
		float tmp = 0;
		
		for (int i = 0; i < k; i++)
		{
			tmp += *C**D;
			C += k;
			D++;
		}
		
		B[y*k + x] -= tmp;
	}
	
	__syncthreads();
    
	copySubMatrix(B, Bglobal, Cm, Dn, k, h);
}

void updateDown(int n, int k, int h, float* Ad)
{
	dim3 grid((n - 1)/k, (n - 1)/k);
	dim3 block(k, k);
	int size = 3*k*k*sizeof(float);
	
	updateDownKernel<<<grid, block, size>>>(n, k, h, Ad);
}

void LU(int n, int k, float* A)
{
	float* Ad;
	HANDLE_ERROR(hipMalloc(&Ad, n*n*sizeof(float)));
	
	syncDevice(A, Ad, n, n);
	
	for (int i = 0; i < n; i += k)
	{
		int w = std::min(k, n - i);
		float* leftTop = Ad + i*n + i;
		
		factorize(n - i, w, n, leftTop);
		
		//if (i < N - 1)
		{
			updateRight(k, n - i, n, leftTop);
			updateDown(n - i, k, n, leftTop);
		}
		
		//printDeviceMatrix(n, n, A, Ad);
	}
	
	syncHost(A, Ad, n, n);
	
	HANDLE_ERROR(hipFree(Ad));
}

// Solve Lx = b for x.
void forwardSubstitution(int n, real* A, real* x, real* b)
{
	for (int i = 0; i < n; ++i)
	{
		real sum = b[i];
		for (int j = 0; j < i; ++j)
			sum -= A(i, j, n)*x[j];
		x[i] = sum;
	}
}

// Solve Ux = b for x.
void backwardSubstitution(int n, real* A, real* x, real* b)
{
	for (int i = n - 1; i >= 0; --i)
	{
		real sum = b[i];
		for (int j = i + 1; j < n; ++j)
			sum -= A(i, j, n)*x[j];
		x[i] = sum/A(i, i, n);
	}
}

int main(int argc, char** argv)
{
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	fprintf(stderr, "# Device name: %s\n", prop.name);

    using namespace std::chrono;

    real* A;
    int n;
    real* b;    
    int k = init(argc, argv, &n, &A, &b, false);
	real* x = new real[n];
	
	k = min(k, 32); // 32*32 is the maximum number of threads per block
	
	auto start = high_resolution_clock::now();
	LU(n, k, A);
	auto end = high_resolution_clock::now();
	
	forwardSubstitution(n, A, x, b);
	backwardSubstitution(n, A, b, x);
	
	//printMatrix(n, n, A);
	
	nanoseconds elapsedTime = end - start;
	printResult(n, b, elapsedTime.count(), 2./3*n*n*n, k, k*k);
	
	delete[] A;
	delete[] x;
	return 0;
}

