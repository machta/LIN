#include "common.h"

#include <algorithm>

#include <hipsolver.h>

#define A(r, c, h) A[(h)*(c) + (r)]
//#define A(r, c) A[n*(c) + (r)]

#define HANDLE_ERROR(err) HandleError(err, __FILE__, __LINE__)

void HandleError(hipError_t err, const char* file, int line)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(1);
	}
}

void LU(int n, real* A)
{
	hipsolverHandle_t handle;
	hipsolverDnCreate(&handle);
	
	float* Ad;
	HANDLE_ERROR(hipMalloc(&Ad, n*n*sizeof(float)));
	
	int Lwork;
	hipsolverDnSgetrf_bufferSize(handle, n, n, Ad, n, &Lwork);	
	float* workspace;
	HANDLE_ERROR(hipMalloc(&workspace, Lwork*sizeof(float)));
	
	int* pivot;
	HANDLE_ERROR(hipMalloc(&pivot, n*sizeof(int)));
	
	int* info;
	HANDLE_ERROR(hipMalloc(&info, sizeof(int)));
	
	HANDLE_ERROR(hipMemcpy(Ad, A, n*n*sizeof(float), hipMemcpyHostToDevice));
	hipsolverDnSgetrf(handle, n, n, Ad, n, workspace, pivot, info);
	HANDLE_ERROR(hipMemcpy(A, Ad, n*n*sizeof(float), hipMemcpyDeviceToHost));
	
	hipsolverDnDestroy(handle);
	HANDLE_ERROR(hipFree(Ad));
	HANDLE_ERROR(hipFree(workspace));
	HANDLE_ERROR(hipFree(pivot));
	HANDLE_ERROR(hipFree(info));
}

// Solve Lx = b for x.
void forwardSubstitution(int n, real* A, real* x, real* b)
{
	for (int i = 0; i < n; ++i)
	{
		real sum = b[i];
		for (int j = 0; j < i; ++j)
			sum -= A(i, j, n)*x[j];
		x[i] = sum;
	}
}

// Solve Ux = b for x.
void backwardSubstitution(int n, real* A, real* x, real* b)
{
	for (int i = n - 1; i >= 0; --i)
	{
		real sum = b[i];
		for (int j = i + 1; j < n; ++j)
			sum -= A(i, j, n)*x[j];
		x[i] = sum/A(i, i, n);
	}
}

int main(int argc, char** argv)
{
    using namespace std::chrono;

    real* A;
    int n;
    real* b;    
    int k = init(argc, argv, &n, &A, &b, false);
	real* x = new real[n];
	
	auto start = high_resolution_clock::now();	
	LU(n, A);	
	auto end = high_resolution_clock::now();
	
	forwardSubstitution(n, A, x, b);
	backwardSubstitution(n, A, b, x);
	
	//printMatrix(n, n, A);
	
	nanoseconds elapsedTime = end - start;
	printResult(n, b, elapsedTime.count(), 2./3*n*n*n, k, 1024);
	
	delete[] A;
	delete[] x;
	return 0;
}

