#include "common.h"

#include <algorithm>

#include <hipblas.h>

#define A(r, c, h) A[(h)*(c) + (r)]
//#define A(r, c) A[n*(c) + (r)]

// Solve Lx = b for x.
void forwardSubstitution(int n, real* A, real* x, real* b)
{
	for (int i = 0; i < n; ++i)
	{
		real sum = b[i];
		for (int j = 0; j < i; ++j)
			sum -= A(i, j, n)*x[j];
		x[i] = sum;
	}
}

// Solve Ux = b for x.
void backwardSubstitution(int n, real* A, real* x, real* b)
{
	for (int i = n - 1; i >= 0; --i)
	{
		real sum = b[i];
		for (int j = i + 1; j < n; ++j)
			sum -= A(i, j, n)*x[j];
		x[i] = sum/A(i, i, n);
	}
}

int main(int argc, char** argv)
{
    using namespace std::chrono;

    real* A;
    int n;
    real* b;    
    int k = init(argc, argv, &n, &A, &b, false);
	real* x = new real[n];
	
	hipError_t error;
	float* Ad;
	int* infod;
	int* pivotd;
	error = hipMalloc((void**) &Ad, n*n*sizeof(real));
	error = hipMalloc((void**) &infod, sizeof(int));
	error = hipMalloc((void**) &pivotd, n*sizeof(int));
	
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	stat = hipblasCreate(&handle);
	
	printMatrix(n, n, A);
	
	auto start = high_resolution_clock::now();
	error = hipMemcpy(Ad, A, n*n*sizeof(real), hipMemcpyHostToDevice);
	
	//for (int i = 0; i < n*n; i++) A[i] = 0;
	
	printMatrix(n, n, A);
	
	stat = hipblasSgetrfBatched(handle, n, &Ad, n, pivotd, infod, 1);
	
	hipMemcpy(A, Ad, n*n*sizeof(real), hipMemcpyDeviceToHost);
	int info = -1000000;
	printf("info = %d\n", info);
	hipMemcpy(&info, infod, sizeof(int), hipMemcpyDeviceToHost);
	printf("info = %d\n", info);
	int* pivot = new int[n];
	hipMemcpy(pivot, pivotd, n*sizeof(int), hipMemcpyDeviceToHost);
	auto end = high_resolution_clock::now();
	
	forwardSubstitution(n, A, x, b);
	backwardSubstitution(n, A, b, x);
	
	printMatrix(n, n, A);
	
	nanoseconds elapsedTime = end - start;
	printResult(n, b, elapsedTime.count(), 2./3*n*n*n, k, 1024);
	
	hipblasDestroy(handle);
	hipFree(Ad);
	hipFree(infod);
	hipFree(pivotd);
	delete[] pivot;
	delete[] A;
	delete[] x;
	return 0;
}

